#include <hip/hip_runtime.h>

// CUDA Kernel for 2D Convolution
__global__ void convolutionKernel(float* input, float* kernel, float* output, 
                                  int inputWidth, int inputHeight, 
                                  int kernelWidth, int kernelHeight) {
    int outX = blockIdx.x * blockDim.x + threadIdx.x;
    int outY = blockIdx.y * blockDim.y + threadIdx.y;

    int outputWidth = inputWidth - kernelWidth + 1;
    int outputHeight = inputHeight - kernelHeight + 1;

    if (outX < outputWidth && outY < outputHeight) {
        float sum = 0.0f;

        for (int ky = 0; ky < kernelHeight; ++ky) {
            for (int kx = 0; kx < kernelWidth; ++kx) {
                int inX = outX + kx;
                int inY = outY + ky;
                sum += input[inY * inputWidth + inX] * kernel[ky * kernelWidth + kx];
            }
        }

        output[outY * outputWidth + outX] = sum;
    }
}

// Wrapper function for launching the kernel
void launchConvolutionKernel(float* input, float* kernel, float* output, 
                             int inputWidth, int inputHeight, 
                             int kernelWidth, int kernelHeight) {
    int outputWidth = inputWidth - kernelWidth + 1;
    int outputHeight = inputHeight - kernelHeight + 1;

    dim3 blockSize(16, 16); // Define block size
    dim3 gridSize((outputWidth + blockSize.x - 1) / blockSize.x,
                  (outputHeight + blockSize.y - 1) / blockSize.y);

    convolutionKernel<<<gridSize, blockSize>>>(input, kernel, output, 
                                               inputWidth, inputHeight, 
                                               kernelWidth, kernelHeight);
    hipDeviceSynchronize(); // Ensure all threads finish execution
}
