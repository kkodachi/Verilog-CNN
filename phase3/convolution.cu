#include <hip/hip_runtime.h>

// CUDA Kernel for 2D Convolution
__global__ void convKernel(float* input, float* kernel, float* output, 
                                  int inputWidth, int inputHeight, 
                                  int kernelWidth, int kernelHeight) {
    // calc thread ind
    int outX = blockIdx.x * blockDim.x + threadIdx.x;
    int outY = blockIdx.y * blockDim.y + threadIdx.y;

    int outputWidth = inputWidth - kernelWidth + 1;
    int outputHeight = inputHeight - kernelHeight + 1;
    // if valid in
    if (outX < outputWidth && outY < outputHeight) {
        float sum = 0.0f;
        // each thread computes its portion of MM
        for (int ky = 0; ky < kernelHeight; ++ky) {
            for (int kx = 0; kx < kernelWidth; ++kx) {
                int inX = outX + kx;
                int inY = outY + ky;
                sum += input[inY * inputWidth + inX] * kernel[ky * kernelWidth + kx];
            }
        }

        output[outY * outputWidth + outX] = sum;
    }
}

// wrapper for launching  kernel
void startConvKernel(float* input, float* kernel, float* output, 
                             int inputWidth, int inputHeight, 
                             int kernelWidth, int kernelHeight) {
    int outputWidth = inputWidth - kernelWidth + 1;
    int outputHeight = inputHeight - kernelHeight + 1;

    dim3 blockSize(16, 16);
    dim3 gridSize((outputWidth + blockSize.x - 1) / blockSize.x,
                  (outputHeight + blockSize.y - 1) / blockSize.y);

    convKernel<<<gridSize, blockSize>>>(input, kernel, output, 
                                               inputWidth, inputHeight, 
                                               kernelWidth, kernelHeight);
    // wait for all threads to finish
    hipDeviceSynchronize();
}
